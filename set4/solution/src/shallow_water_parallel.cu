#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>

#include "../inc/argument_utils.h"

typedef int64_t int_t;
typedef double real_t;

int_t
        N,
        max_iteration,
        snapshot_frequency;

const real_t
        domain_size = 10.0,
        gravity = 9.81,
        density = 997.0;

// TODO #1.1: Declare device pointers
real_t
//*h_mass[2] = { NULL, NULL },
*h_mass_0 = NULL,
        *h_mass_1 = NULL,
//*h_mass_velocity_x[2] = { NULL, NULL },
*h_mass_velocity_x_0 = NULL,
        *h_mass_velocity_x_1 = NULL,
        *h_velocity_x = NULL,
        *h_acceleration_x = NULL,
        dx,
        dt;

real_t
        *d_mass_0 = NULL,
        *d_mass_1 = NULL,
        *d_mass_velocity_x_0 = NULL,
        *d_mass_velocity_x_1 = NULL,
        *d_velocity_x = NULL,
        *d_acceleration_x = NULL;


#define PN(x)        mass_0[(x)]
#define PN_next(x)   mass_1[(x)]
#define PNU(x)       mass_velocity_x_0[(x)]
#define PNU_next(x)  mass_velocity_x_1[(x)]
#define U(x)         velocity_x[(x)]
#define DU(x)        acceleration_x[(x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *velocity_x, int_t N);

__global__ void time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0,
            real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N);

__device__ void boundary_condition(real_t *domain_variable, int sign);

void domain_init(void);

void domain_save(int_t iteration);

void domain_finalize(void);


void swap(real_t **m1, real_t **m2)
{
    real_t *tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}


int main(int argc, char **argv)
{
    OPTIONS *options = parse_args(argc, argv);
    if (!options)
    {
        fprintf(stderr, "Argument parsing failed\n");
        exit(1);
    }
    
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;
    
    domain_init();
    
    // TODO #2.4: Define the thread block size and calculate the corresponding grid size.
    
    dim3 threadBlockDims = {1024, 1, 1};
    dim3 gridDims = {N / 1024 + ((N % 1024) != 0), 1, 1};
    
    for (int_t iteration = 0; iteration <= max_iteration; iteration++)
    {
        // TODO #1.5: Move the boundary_condition calls to inside the time_step_1 kernel
        //boundary_condition(h_mass_0, 1);
        //boundary_condition(h_mass_velocity_x_0, -1);
        
        // TODO #1.6: Change the function calls to invoke the kernels with the arguments to the device-side equivalents
        time_step_1<<<gridDims, threadBlockDims>>>(d_acceleration_x, d_mass_0, d_velocity_x, N);
        time_step_2<<<gridDims, threadBlockDims>>>(d_mass_velocity_x_0, d_mass_velocity_x_1, d_acceleration_x, d_mass_0,
                                                   d_mass_1, d_velocity_x, dx, dt, N);
        
        if (iteration % snapshot_frequency == 0)
        {
            printf(
                    "Iteration %ld of %ld (%.2lf%% complete)\n",
                    iteration,
                    max_iteration,
                    100.0 * (real_t) iteration / (real_t) max_iteration
            );
            
            // TODO #1.8: Copy the results back from the GPU into the host-side buffers.
    
            //****************** CUDA ******************
    
            hipMemcpy(h_mass_0, d_mass_0, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_1, d_mass_1, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
    
            hipMemcpy(h_mass_velocity_x_0, d_mass_velocity_x_0, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_x_1, d_mass_velocity_x_1, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
    
            hipMemcpy(h_velocity_x, d_velocity_x, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_acceleration_x, d_acceleration_x, (N + 2) * sizeof(real_t), hipMemcpyDeviceToHost);
    
            //****************** CUDA ******************
            
            domain_save(iteration);
        }
        
        // TODO #1.7: Swap device pointers
        swap(&h_mass_0, &h_mass_1);
        swap(&h_mass_velocity_x_0, &h_mass_velocity_x_1);
    
        swap(&d_mass_0, &d_mass_1);
        swap(&d_mass_velocity_x_0, &d_mass_velocity_x_1);
    }
    
    domain_finalize();
    
    exit(EXIT_SUCCESS);
}

// TODO #1.5: Change the host-side function to be a device-side function
__device__ void boundary_condition(real_t *domain_variable, int sign, int_t N, bool last)
{
#define VAR(x) domain_variable[(x)]
    //if (!last)
        VAR(0) = sign * VAR(2);
    //else
        VAR(N + 1) = sign * VAR(N - 1);
#undef VAR
}

// TODO #1.4: Change the function to be a CUDA kernel
__global__ void time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *velocity_x, int_t N)
{
    
    // TODO #2.1: Define the global index
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= N)
        return;
    
    // TODO #2.3: Restrict the boundary_condition updates to only be performed by the first and last thread
    if (global_index == 0 || global_index == N - 1)
        boundary_condition(mass_0, 1, N, global_index);
    
    if (global_index == 0)
        DU(global_index) = PN(global_index) * U(global_index) * U(global_index)
                           + 0.5 * gravity * PN(global_index) * PN(global_index) / density;
    else if (global_index == N - 1)
        DU(global_index + 2) = PN(global_index + 2) * U(global_index + 2) * U(global_index + 2)
                           + 0.5 * gravity * PN(global_index + 2) * PN(global_index + 2) / density;
    /*
    for (int_t x = 0; x <= N + 1; x++)
    {*/
    int_t x = global_index + 1; //This takes care of 1 -> N
        DU(x) = PN(x) * U(x) * U(x)
                + 0.5 * gravity * PN(x) * PN(x) / density;
    /*}*/
}

// TODO #1.4: Change the function to be a CUDA kernel
__global__ void
time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0,
            real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N)
{
    
    // TODO #2.1: Define the global index
    int global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index >= N)
        return;
    
    if (global_index == 0 || global_index == N - 1)
        boundary_condition(mass_0, 1, N, global_index);
    
    /*
    for (int_t x = 1; x <= N; x++)
    {*/
    int_t x = global_index + 1;
        PNU_next(x) = 0.5 * (PNU(x + 1) + PNU(x - 1)) - dt * (
                (DU(x + 1) - DU(x - 1)) / (2 * dx)
        );
    /*}*/
    /*
    for (int_t x = 1; x <= N; x++)
    {*/
        PN_next(x) = 0.5 * (PN(x + 1) + PN(x - 1)) - dt * (
                (PNU(x + 1) - PNU(x - 1)) / (2 * dx)
        );
    /*}*/
    /*
    for (int_t x = 1; x <= N; x++)
    {*/
        U(x) = PNU_next(x) / PN_next(x);
    /*}*/
}


void domain_init(void)
{
    // TODO #1.2: Allocate space for the device-side buffers.
    
    h_mass_0 = (real_t *) calloc((N + 2), sizeof(real_t));
    h_mass_1 = (real_t *) calloc((N + 2), sizeof(real_t));
    
    h_mass_velocity_x_0 = (real_t *) calloc((N + 2), sizeof(real_t));
    h_mass_velocity_x_1 = (real_t *) calloc((N + 2), sizeof(real_t));
    
    h_velocity_x = (real_t *) calloc((N + 2), sizeof(real_t));
    h_acceleration_x = (real_t *) calloc((N + 2), sizeof(real_t));
    
    //****************** CUDA ******************
    
    hipMalloc(&d_mass_0, (N + 2) * sizeof(real_t));
    hipMalloc(&d_mass_1, (N + 2) * sizeof(real_t));
    
    hipMalloc(&d_mass_velocity_x_0, (N + 2) * sizeof(real_t));
    hipMalloc(&d_mass_velocity_x_1, (N + 2) * sizeof(real_t));
    
    hipMalloc(&d_velocity_x, (N + 2) * sizeof(real_t));
    hipMalloc(&d_acceleration_x, (N + 2) * sizeof(real_t));
    
    //****************** CUDA ******************
    
    
    // Data initialization
    for (int_t x = 1; x <= N; x++)
    {
        h_mass_0[(x)] = 1e-3;
        h_mass_velocity_x_0[(x)] = 0.0;
        
        real_t c = x - (double) N / 2;
        if (sqrt(c * c) < N / 20.0)
        {
            h_mass_0[(x)] -= 5e-4 * exp(-4 * pow(c, 2.0) / (real_t) (N));
        }
        
        h_mass_0[(x)] *= density;
    }
    
    dx = domain_size / (real_t) N;
    dt = 0.1 * dx;
    
    // TODO #1.3: Transfer the host-side data to the device.
    
    //****************** CUDA ******************
    
    hipMemcpy(d_mass_0, h_mass_0, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_1, h_mass_1, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x_0, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x_1, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_velocity_x, h_velocity_x, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, h_acceleration_x, (N + 2) * sizeof(real_t), hipMemcpyHostToDevice);
    
    //****************** CUDA ******************
}


void domain_save(int_t iteration)
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset(filename, 0, 256 * sizeof(char));
    sprintf(filename, "data/%.5ld.bin", index);
    
    FILE *out = fopen(filename, "wb");
    if (!out)
    {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    fwrite(&h_mass_0[1], sizeof(real_t), N, out);
    fclose(out);
}


void domain_finalize(void)
{
    free(h_mass_0);
    free(h_mass_1);
    free(h_mass_velocity_x_0);
    free(h_mass_velocity_x_1);
    free(h_velocity_x);
    free(h_acceleration_x);
    
    // TODO #1.9: Add hipFree for the device buffers.
    
    hipFree(d_mass_0);
    hipFree(d_mass_1);
    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_velocity_x);
    hipFree(d_acceleration_x);
}
