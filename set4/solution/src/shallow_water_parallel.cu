#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>

#include "../inc/argument_utils.h"

typedef int64_t int_t;
typedef double real_t;

int_t
    N,
    max_iteration,
    snapshot_frequency;

const real_t
    domain_size = 10.0,
    gravity = 9.81,
    density = 997.0;

// TODO #1.1: Declare device pointers
real_t
    *h_mass[2] = { NULL, NULL },
    *h_mass_velocity_x[2] = { NULL, NULL },
    *h_velocity_x = NULL,
    *h_acceleration_x = NULL,
    dx,
    dt;

#define PN(x)        mass_0[(x)]
#define PN_next(x)   mass_1[(x)]
#define PNU(x)       mass_velocity_x_0[(x)]
#define PNU_next(x)  mass_velocity_x_1[(x)]
#define U(x)         velocity_x[(x)]
#define DU(x)        acceleration_x[(x)]

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }                 
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)  
{                                                                                     
    if (code != hipSuccess) {                                                        
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);                                                        
    }                                                                                 
}

void time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *velocity_x, int_t N);
void time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0, real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N);
void boundary_condition( real_t *domain_variable, int sign );
void domain_init ( void );
void domain_save ( int_t iteration );
void domain_finalize ( void );


void
swap ( real_t** m1, real_t** m2 )
{
    real_t* tmp;
    tmp = *m1;
    *m1 = *m2;
    *m2 = tmp;
}


int
main ( int argc, char **argv )
{
    OPTIONS *options = parse_args( argc, argv );
    if ( !options )
    {
        fprintf( stderr, "Argument parsing failed\n" );
        exit(1);
    }

    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;

    domain_init();

    // TODO #2.4: Define the thread block size and calculate the corresponding grid size.


    for ( int_t iteration = 0; iteration <= max_iteration; iteration++ )
    {
        // TODO #1.5: Move the boundary_condition calls to inside the time_step_1 kernel
        boundary_condition(h_mass[0], 1);
        boundary_condition(h_mass_velocity_x[0], -1);

        // TODO #1.6: Change the function calls to invoke the kernels with the arguments to the device-side equivalents
        time_step_1(h_acceleration_x, h_mass[0], h_velocity_x, N);
        time_step_2(h_mass_velocity_x[0], h_mass_velocity_x[1], h_acceleration_x, h_mass[0], h_mass[1], h_velocity_x, dx, dt, N);

        if ( iteration % snapshot_frequency == 0 )
        {
            printf (
                "Iteration %ld of %ld (%.2lf%% complete)\n",
                iteration,
                max_iteration,
                100.0 * (real_t) iteration / (real_t) max_iteration
            );

            // TODO #1.8: Copy the results back from the GPU into the host-side buffers.

            domain_save ( iteration );
        }

        // TODO #1.7: Swap device pointers
        swap( &h_mass[0], &h_mass[1] );
        swap( &h_mass_velocity_x[0], &h_mass_velocity_x[1] );
    }

    domain_finalize();

    exit ( EXIT_SUCCESS );
}

// TODO #1.5: Change the host-side function to be a device-side function
void
boundary_condition ( real_t *domain_variable, int sign )
{
    #define VAR(x) domain_variable[(x)]
    VAR(   0 ) = sign*VAR( 2   );
    VAR( N+1 ) = sign*VAR( N-1 );
    #undef VAR
}

// TODO #1.4: Change the function to be a CUDA kernel
void
time_step_1(real_t *acceleration_x, real_t *mass_0, real_t *velocity_x, int_t N)
{
    // TODO #2.1: Define the global index
    // TODO #2.3: Restrict the boundary_condition updates to only be performed by the first and last thread
    for ( int_t x=0; x<=N+1; x++ )
    {
        DU(x) = PN(x) * U(x) * U(x)
                + 0.5 * gravity * PN(x) * PN(x) / density;
    }
}

// TODO #1.4: Change the function to be a CUDA kernel
void
time_step_2(real_t *mass_velocity_x_0, real_t *mass_velocity_x_1, real_t *acceleration_x, real_t *mass_0, real_t *mass_1, real_t *velocity_x, real_t dx, real_t dt, int_t N)
{
    // TODO #2.1: Define the global index
    for ( int_t x=1; x<=N; x++ )
    {
        PNU_next(x) = 0.5*( PNU(x+1) + PNU(x-1) ) - dt*(
                      ( DU(x+1) - DU(x-1) ) / (2*dx)
        );
    }

    for ( int_t x=1; x<=N; x++ )
    {
        PN_next(x) = 0.5*( PN(x+1) + PN(x-1) ) - dt*(
                       ( PNU(x+1) - PNU(x-1) ) / (2*dx)
        );
    }

    for ( int_t x=1; x<=N; x++ )
    {
        U(x) = PNU_next(x) / PN_next(x);
    }
}


void
domain_init ( void )
{
    // TODO #1.2: Allocate space for the device-side buffers.

    h_mass[0] = (real_t *) calloc ( (N+2), sizeof(real_t) );
    h_mass[1] = (real_t *) calloc ( (N+2),  sizeof(real_t) );

    h_mass_velocity_x[0] = (real_t *) calloc ( (N+2), sizeof(real_t) );
    h_mass_velocity_x[1] = (real_t *) calloc ( (N+2),  sizeof(real_t) );

    h_velocity_x = (real_t *) calloc ( (N+2), sizeof(real_t) );
    h_acceleration_x = (real_t *) calloc ( (N+2), sizeof(real_t) );

    // Data initialization
    for ( int_t x=1; x<=N; x++ )
    {
        h_mass[0][(x)] = 1e-3;
        h_mass_velocity_x[0][(x)] = 0.0;

        real_t c = x-(double)N/2;
        if ( sqrt ( c*c ) < N/20.0 )
        {
            h_mass[0][(x)] -= 5e-4 * exp(-4 * pow(c, 2.0) / (real_t) (N));
        }

        h_mass[0][(x)] *= density;
    }

    dx = domain_size / (real_t) N;
    dt = 0.1*dx;

    // TODO #1.3: Transfer the host-side data to the device.

}


void
domain_save ( int_t iteration )
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset ( filename, 0, 256*sizeof(char) );
    sprintf ( filename, "data/%.5ld.bin", index );

    FILE *out = fopen ( filename, "wb" );
    if ( ! out ) {
        fprintf(stderr, "Failed to open file: %s\n", filename);
        exit(1);
    }
    fwrite( &h_mass[0][1], sizeof(real_t), N, out );
    fclose ( out );
}


void
domain_finalize ( void )
{
    free ( h_mass[0] );
    free ( h_mass[1] );
    free ( h_mass_velocity_x[0] );
    free ( h_mass_velocity_x[1] );
    free ( h_velocity_x );
    free ( h_acceleration_x );

    // TODO #1.9: Add hipFree for the device buffers.
}
