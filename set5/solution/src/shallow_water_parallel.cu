#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// TDT4200 Parallel Computing - Graded CUDA
// ---------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>

#include <iostream>
#include <hip/hip_cooperative_groups.h>

#include "../inc/argument_utils.h"


typedef int64_t int_t;
typedef double real_t;

int_t
        N,
        max_iteration,
        snapshot_frequency;

const real_t
        domain_size = 10.0,
        gravity = 9.81,
        density = 997.0;

real_t
        *h_mass_0 = NULL,
        *h_mass_1 = NULL,
        *d_mass_0 = NULL,
        *d_mass_1 = NULL,
        
        *h_mass_velocity_x_0 = NULL,
        *h_mass_velocity_x_1 = NULL,
        *d_mass_velocity_x_0 = NULL,
        *d_mass_velocity_x_1 = NULL,
        
        *h_mass_velocity_y_0 = NULL,
        *h_mass_velocity_y_1 = NULL,
        *d_mass_velocity_y_0 = NULL,
        *d_mass_velocity_y_1 = NULL,
        
        *h_mass_velocity = NULL,
        *d_mass_velocity = NULL,
        
        *h_velocity_x = NULL,
        *d_velocity_x = NULL,
        *h_velocity_y = NULL,
        *d_velocity_y = NULL,
        
        *h_acceleration_x = NULL,
        *d_acceleration_x = NULL,
        *h_acceleration_y = NULL,
        *d_acceleration_y = NULL,
        dx,
        dt;

#define PN(y, x)         mass_0[(y)*(N+2)+(x)]
#define PN_next(y, x)    mass_1[(y)*(N+2)+(x)]
#define PNU(y, x)        mass_velocity_x_0[(y)*(N+2)+(x)]
#define PNU_next(y, x)   mass_velocity_x_1[(y)*(N+2)+(x)]
#define PNV(y, x)        mass_velocity_y_0[(y)*(N+2)+(x)]
#define PNV_next(y, x)   mass_velocity_y_1[(y)*(N+2)+(x)]
#define PNUV(y, x)       mass_velocity[(y)*(N+2)+(x)]
#define U(y, x)          velocity_x[(y)*(N+2)+(x)]
#define V(y, x)          velocity_y[(y)*(N+2)+(x)]
#define DU(y, x)         acceleration_x[(y)*(N+2)+(x)]
#define DV(y, x)         acceleration_y[(y)*(N+2)+(x)]


#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*
void time_step(real_t *velocity_x, real_t *velocity_y,
               real_t *acceleration_x, real_t *acceleration_y,
               real_t *mass_velocity_x_0, real_t *mass_velocity_x_1,
               real_t *mass_velocity_y_0, real_t *mass_velocity_y_1,
               real_t *mass_velocity, real_t *mass_0, real_t *mass_1
);*/

void __global__ time_step(real_t *mass_0,
                          real_t *mass_1,
                          real_t *mass_velocity_x_0,
                          real_t *mass_velocity_x_1,
                          real_t *mass_velocity_y_0,
                          real_t *mass_velocity_y_1,
                          real_t *mass_velocity,
                          real_t *velocity_x,
                          real_t *velocity_y,
                          real_t *acceleration_x,
                          real_t *acceleration_y,
                          int_t N,
                          real_t dx,
                          real_t dt);

void __global__ time_step_1(real_t *mass_0,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity,
                            real_t *velocity_x,
                            real_t *velocity_y,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N);

void __global__ time_step_2(real_t *mass_0,
                            real_t *mass_1,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_x_1,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity_y_1,
                            real_t *mass_velocity,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N,
                            real_t dx,
                            real_t dt);

// TODO: Rewrite boundary_condition as a device function.
void __device__ boundary_condition(real_t *domain_variable, int sign, int_t N, int_t x_idx, int_t y_idx);

void __device__ boundary_condition_advanced(real_t *domain_variable, int sign, int_t N);

void domain_init(void);

void domain_save(int_t iteration);

void domain_finalize(void);

// Pthreads threaded domain save function
void *domain_save_threaded(void *iter);

void swap(real_t **t1, real_t **t2)
{
    real_t *tmp;
    tmp = *t1;
    *t1 = *t2;
    *t2 = tmp;
}


typedef struct time_step_data
{
    real_t *mass_0;
    real_t *mass_1;
    real_t *mass_velocity_x_0;
    real_t *mass_velocity_x_1;
    real_t *mass_velocity_y_0;
    real_t *mass_velocity_y_1;
    real_t *mass_velocity;
    real_t *velocity_x;
    real_t *velocity_y;
    real_t *acceleration_x;
    real_t *acceleration_y;
    int_t N;
    real_t dx;
    real_t dt;
    
    
    time_step_data(real_t *a_mass_0,
                   real_t *a_mass_1,
                   real_t *a_mass_velocity_x_0,
                   real_t *a_mass_velocity_x_1,
                   real_t *a_mass_velocity_y_0,
                   real_t *a_mass_velocity_y_1,
                   real_t *a_mass_velocity,
                   real_t *a_velocity_x,
                   real_t *a_velocity_y,
                   real_t *a_acceleration_x,
                   real_t *a_acceleration_y,
                   int_t a_N,
                   real_t a_dx,
                   real_t a_dt) :
            mass_0(a_mass_0),
            mass_1(a_mass_1),
            mass_velocity_x_0(a_mass_velocity_x_0),
            mass_velocity_x_1(a_mass_velocity_x_1),
            mass_velocity_y_0(a_mass_velocity_y_0),
            mass_velocity_y_1(a_mass_velocity_y_1),
            mass_velocity(a_mass_velocity),
            velocity_x(a_velocity_x),
            velocity_y(a_velocity_y),
            acceleration_x(a_acceleration_x),
            acceleration_y(a_acceleration_y),
            N(a_N),
            dx(a_dx),
            dt(a_dt)
    {}
    
} time_step_data_t;


int main(int argc, char **argv)
{
    
    OPTIONS *options = parse_args(argc, argv);
    if (!options)
    {
        fprintf(stderr, "Argument parsing failed\n");
        exit(1);
    }
    
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;
    
    domain_init();
    
    //Parallel CUDA
    //We want 1024 threads per block (maximum) for a square this is 32x32 = 1024
    //Hence we have 32 threads in x and y dim, in thread block
    dim3 threadBlockDims = {32, 32, 1};
    //We want to divide the 2D space into equal sized squares making a grid.
    //This grid then needs to be N / 32(number of threads in dim in thread block) (rounded up).
    //Therefore, we get ceiling(N / threadBlock.x) and corresponding for y-dim.
    dim3 gridDims = {(uint32_t)((N + 2) / threadBlockDims.x) + (((N + 2) % threadBlockDims.x) != 0),
                     (uint32_t)((N + 2) / threadBlockDims.y) + (((N + 2) % threadBlockDims.y) != 0),
                     1};
    
    //Advanced CUDA
    int THREADS; //Number of threads per block recommended by the hardware
    int BLOCKS; //Number of blocks that can run at once on hardware
    hipOccupancyMaxPotentialBlockSize(&BLOCKS, &THREADS, time_step, 0, 0);
    std::cout << THREADS << std::endl;
    std::cout << BLOCKS << std::endl;
    
    //Get the biggest square from the number of threads possible per block.
    //Will most likely be 32x32.
    threadBlockDims = {(uint32_t) sqrt(THREADS), (uint32_t) sqrt(THREADS), 1};
    //We only have BLOCKS number og blocks that fit on the device at a time
    //We need to ensure that everything is on the device at the same time
    //for the sync to function, and not deadlock.
    //Therefore, we have the biggest square the that allows the number of
    //BLOCKS we have available. Threads may have to do multiple nodes
    //in the calculation to compensate for the bound number of BLOCKS.
    gridDims = {(uint32_t) sqrt(BLOCKS), (uint32_t) sqrt(BLOCKS), 1};
    
    for (int_t iteration = 0; iteration <= max_iteration; iteration++)
    {
        //From serial code
        /*
        time_step(h_velocity_x, h_velocity_y,
                  h_acceleration_x, h_acceleration_y,
                  h_mass_velocity_x_0, h_mass_velocity_x_1,
                  h_mass_velocity_y_0, h_mass_velocity_y_1,
                  h_mass_velocity, h_mass_0, h_mass_1
        );*/
        
        // TODO: Launch time_step kernels
        //This is the code for the parallel CUDA
        /*
        time_step_1<<<gridDims, threadBlockDims>>>(d_mass_0,
                                                   d_mass_velocity_x_0,
                                                   d_mass_velocity_y_0,
                                                   d_mass_velocity,
                                                   d_velocity_x,
                                                   d_velocity_y,
                                                   d_acceleration_x,
                                                   d_acceleration_y,
                                                   N);
        
        hipDeviceSynchronize();
        time_step_2<<<gridDims, threadBlockDims>>>(d_mass_0,
                                                   d_mass_1,
                                                   d_mass_velocity_x_0,
                                                   d_mass_velocity_x_1,
                                                   d_mass_velocity_y_0,
                                                   d_mass_velocity_y_1,
                                                   d_mass_velocity,
                                                   d_acceleration_x,
                                                   d_acceleration_y,
                                                   N,
                                                   dx,
                                                   dt);
        */
        //This is the code for the advanced CUDA
        void *kernel_args[] = {(void *) &d_mass_0,
                               (void *) &d_mass_1,
                               (void *) &d_mass_velocity_x_0,
                               (void *) &d_mass_velocity_x_1,
                               (void *) &d_mass_velocity_y_0,
                               (void *) &d_mass_velocity_y_1,
                               (void *) &d_mass_velocity,
                               (void *) &d_velocity_x,
                               (void *) &d_velocity_y,
                               (void *) &d_acceleration_x,
                               (void *) &d_acceleration_y,
                               (void *) &N,
                               (void *) &dx,
                               (void *) &dt};
        
        hipLaunchCooperativeKernel((void *) time_step, gridDims, threadBlockDims, kernel_args);
        
        if (iteration % snapshot_frequency == 0)
        {
            printf(
                    "Iteration %ld of %ld, (%.2lf%% complete)\n",
                    iteration,
                    max_iteration,
                    100.0 * (real_t) iteration / (real_t) max_iteration
            );
            
            
            // TODO: Copy the masses from the device to host prior to domain_save
            //Transfer the device-side data to the host
            //****************** CUDA ******************
            int elements = (N + 2) * (N + 2);
            
            hipMemcpy(h_mass_0, d_mass_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            /* These are not needed, we only use mass_0 to write to file
            hipMemcpy(h_mass_1, d_mass_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_x_0, d_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_x_1, d_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_y_0, d_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_y_1, d_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(h_mass_velocity, d_mass_velocity, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(h_velocity_x, d_velocity_x, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_velocity_y, d_velocity_y, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_acceleration_x, d_acceleration_x, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_acceleration_y, d_acceleration_y, elements * sizeof(real_t), hipMemcpyDeviceToHost);
             */
            //****************** CUDA ******************
            
            domain_save(iteration);
        }
        
        // TODO: Swap device buffer pointers between iterations
        swap(&d_mass_0, &d_mass_1);
        swap(&d_mass_velocity_x_0, &d_mass_velocity_x_1);
        swap(&d_mass_velocity_y_0, &d_mass_velocity_y_1);
        
        swap(&h_mass_0, &h_mass_1);
        swap(&h_mass_velocity_x_0, &h_mass_velocity_x_1);
        swap(&h_mass_velocity_y_0, &h_mass_velocity_y_1);
    }
    
    domain_finalize();
    
    exit(EXIT_SUCCESS);
}

//Old serial time step
// TODO: Rewrite this function as one or more CUDA kernels
// ---------------------------------------------------------
// To ensure correct results, the participating threads in the thread
// grid must be synchronized after calculating the accelerations (DU, DV).
// If the grid is not synchronized, data dependencies cannot be guaranteed.
/*
void time_step(real_t *velocity_x, real_t *velocity_y,
               real_t *acceleration_x, real_t *acceleration_y,
               real_t *mass_velocity_x_0, real_t *mass_velocity_x_1,
               real_t *mass_velocity_y_0, real_t *mass_velocity_y_1,
               real_t *mass_velocity, real_t *mass_0, real_t *mass_1)
{
    boundary_condition(mass_0, 1);
    boundary_condition(mass_velocity_x_0, -1);
    boundary_condition(mass_velocity_y_0, -1);
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            U(y, x) = PNU(y, x) / PN(y, x);
            V(y, x) = PNV(y, x) / PN(y, x);
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNUV(y, x) = PN(y, x) * U(y, x) * V(y, x);
        }
    
    for (int_t y = 0; y <= N + 1; y++)
        for (int_t x = 0; x <= N + 1; x++)
        {
            DU(y, x) = PN(y, x) * U(y, x) * U(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
            DV(y, x) = PN(y, x) * V(y, x) * V(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNU_next(y, x) = 0.5 * (PNU(y, x + 1) + PNU(y, x - 1)) - dt * (
                    (DU(y, x + 1) - DU(y, x - 1)) / (2 * dx)
                    + (PNUV(y, x + 1) - PNUV(y, x - 1)) / (2 * dx)
            );
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNV_next(y, x) = 0.5 * (PNV(y + 1, x) + PNV(y - 1, x)) - dt * (
                    (DV(y + 1, x) - DV(y - 1, x)) / (2 * dx)
                    + (PNUV(y + 1, x) - PNUV(y - 1, x)) / (2 * dx)
            );
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PN_next(y, x) = 0.25 * (PN(y, x + 1) + PN(y, x - 1) + PN(y + 1, x) + PN(y - 1, x)) - dt * (
                    (PNU(y, x + 1) - PNU(y, x - 1)) / (2 * dx)
                    + (PNV(y + 1, x) - PNV(y - 1, x)) / (2 * dx)
            );
        }
}*/

//Advanced CUDA time step
void __global__ time_step(real_t *mass_0,
                          real_t *mass_1,
                          real_t *mass_velocity_x_0,
                          real_t *mass_velocity_x_1,
                          real_t *mass_velocity_y_0,
                          real_t *mass_velocity_y_1,
                          real_t *mass_velocity,
                          real_t *velocity_x,
                          real_t *velocity_y,
                          real_t *acceleration_x,
                          real_t *acceleration_y,
                          int_t N,
                          real_t dx,
                          real_t dt)
{
    cooperative_groups::grid_group grid = cooperative_groups::this_grid();
    
    int global_index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_index_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    int x_dim_stride = gridDim.x * blockDim.x;
    int y_dim_stride = gridDim.y * blockDim.y;
    
    boundary_condition_advanced(mass_0, 1, N);
    boundary_condition_advanced(mass_velocity_x_0, -1, N);
    boundary_condition_advanced(mass_velocity_y_0, -1, N);
    
    for (int_t y = global_index_y; y <= N; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N; x += x_dim_stride)
        {
            if (x == 0 || y == 0)
                continue;
            U(y, x) = PNU(y, x) / PN(y, x);
            V(y, x) = PNV(y, x) / PN(y, x);
        }
    
    for (int_t y = global_index_y; y <= N; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N; x += x_dim_stride)
        {
            if (x == 0 || y == 0)
                continue;
            PNUV(y, x) = PN(y, x) * U(y, x) * V(y, x);
        }
    
    for (int_t y = global_index_y; y <= N + 1; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N + 1; x += x_dim_stride)
        {
            DU(y, x) = PN(y, x) * U(y, x) * U(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
            DV(y, x) = PN(y, x) * V(y, x) * V(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
        }
    
    grid.sync();
    
    for (int_t y = global_index_y; y <= N; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N; x += x_dim_stride)
        {
            if (x == 0 || y == 0)
                continue;
            PNU_next(y, x) = 0.5 * (PNU(y, x + 1) + PNU(y, x - 1)) - dt * (
                    (DU(y, x + 1) - DU(y, x - 1)) / (2 * dx)
                    + (PNUV(y, x + 1) - PNUV(y, x - 1)) / (2 * dx)
            );
        }
    
    for (int_t y = global_index_y; y <= N; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N; x += x_dim_stride)
        {
            if (x == 0 || y == 0)
                continue;
            PNV_next(y, x) = 0.5 * (PNV(y + 1, x) + PNV(y - 1, x)) - dt * (
                    (DV(y + 1, x) - DV(y - 1, x)) / (2 * dx)
                    + (PNUV(y + 1, x) - PNUV(y - 1, x)) / (2 * dx)
            );
        }
    
    for (int_t y = global_index_y; y <= N; y += y_dim_stride)
        for (int_t x = global_index_x; x <= N; x += x_dim_stride)
        {
            if (x == 0 || y == 0)
                continue;
            PN_next(y, x) = 0.25 * (PN(y, x + 1) + PN(y, x - 1) + PN(y + 1, x) + PN(y - 1, x)) - dt * (
                    (PNU(y, x + 1) - PNU(y, x - 1)) / (2 * dx)
                    + (PNV(y + 1, x) - PNV(y - 1, x)) / (2 * dx)
            );
        }
}

void __global__ time_step_1(real_t *mass_0,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity,
                            real_t *velocity_x,
                            real_t *velocity_y,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N)
{
    int global_index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (global_index_x >= N + 2 || global_index_y >= N + 2)
        return;
    
    
    int_t y = global_index_y;
    int_t x = global_index_x;
    
    boundary_condition(mass_0, 1, N, x, y);
    boundary_condition(mass_velocity_x_0, -1, N, x, y);
    boundary_condition(mass_velocity_y_0, -1, N, x, y);
    
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    U(y, x) = PNU(y, x) / PN(y, x);
    V(y, x) = PNV(y, x) / PN(y, x);
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNUV(y, x) = PN(y, x) * U(y, x) * V(y, x);
    /*}*/
    /*
    for (int_t y = 0; y <= N + 1; y++)
        for (int_t x = 0; x <= N + 1; x++)
        {*/
    DU(y, x) = PN(y, x) * U(y, x) * U(y, x) + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
    DV(y, x) = PN(y, x) * V(y, x) * V(y, x) + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
    /*}*/
}

void __global__ time_step_2(real_t *mass_0,
                            real_t *mass_1,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_x_1,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity_y_1,
                            real_t *mass_velocity,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N,
                            real_t dx,
                            real_t dt)
{
    int global_index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (global_index_x >= N + 2 || global_index_y >= N + 2)
        return;
    
    
    int_t y = global_index_y;
    int_t x = global_index_x;
    
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNU_next(y, x) = 0.5 * (PNU(y, x + 1) + PNU(y, x - 1)) -
                     dt * ((DU(y, x + 1) - DU(y, x - 1)) / (2 * dx) + (PNUV(y, x + 1) - PNUV(y, x - 1)) / (2 * dx));
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNV_next(y, x) = 0.5 * (PNV(y + 1, x) + PNV(y - 1, x)) -
                     dt * ((DV(y + 1, x) - DV(y - 1, x)) / (2 * dx) + (PNUV(y + 1, x) - PNUV(y - 1, x)) / (2 * dx));
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PN_next(y, x) = 0.25 * (PN(y, x + 1) + PN(y, x - 1) + PN(y + 1, x) + PN(y - 1, x)) -
                    dt * ((PNU(y, x + 1) - PNU(y, x - 1)) / (2 * dx) + (PNV(y + 1, x) - PNV(y - 1, x)) / (2 * dx));
    /*}*/
}

//Parallel CUDA code
// TODO: Rewrite boundary_condition as a device function.
void __device__ boundary_condition(real_t *domain_variable, int sign, int_t N, int_t x_idx, int_t y_idx)
{
#define VAR(y, x) domain_variable[(y)*(N+2)+(x)]
    //The corners can't be run in parallel, so it does not
    //matter if we check them with if-statements, all other
    //threads will either stall or do the same operation.
    //Nonetheless, it will take 4 cycles (of writing data).
    //if (x_idx == 0 && y_idx == 0)
    VAR(0, 0) = sign * VAR(2, 2);
    //if (x_idx == 0 && y_idx == N + 1)
    VAR(N + 1, 0) = sign * VAR(N - 1, 2);
    //if (x_idx == N - 1 && y_idx == 0)
    VAR(0, N + 1) = sign * VAR(2, N - 1);
    //if (x_idx == N - 1 && y_idx == N + 1)
    VAR(N + 1, N + 1) = sign * VAR(N - 1, N - 1);
    
    //Each of the boundary sides will be executed in parallel.
    //E.g. every thread with x-index equal 0 will calculate the
    //value for the boundary at side along x = 0, at the y-index
    //the thread has.
    //
    //This is a good speedup since before every thread calculated these values,
    //So it would take N * 4 (sides) cycles to compute, now it takes on side
    //in parallel in one cycle, meaning it in total takes 4 cycles. So it now
    //takes the same amount of time as the corner cases.
    if (x_idx == 0)
        //for (int_t y = 1; y <= N; y++)
        VAR(y_idx, 0) = sign * VAR(y_idx, 2);
    if (x_idx == N + 1)
        //for (int_t y = 1; y <= N; y++)
        VAR(y_idx, N + 1) = sign * VAR(y_idx, N - 1);
    if (y_idx == 0)
        //for (int_t x = 1; x <= N; x++)
        VAR(0, x_idx) = sign * VAR(2, x_idx);
    if (y_idx == N + 1)
        //for (int_t x = 1; x <= N; x++)
        VAR(N + 1, x_idx) = sign * VAR(N - 1, x_idx);

#undef VAR
}

//Advanced CUDA code
void __device__ boundary_condition_advanced(real_t *domain_variable, int sign, int_t N)
{
    //Get the global index values in x- and y-dim.
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    
    //Find the potential striding length in both dims.
    //We want these values since the boundary will be handled
    //by a single call and therefore needs to include the
    //striding that potentially will be needed, we have too few
    //blocks for our work.
    int x_dim_stride = gridDim.x * blockDim.x;
    int y_dim_stride = gridDim.y * blockDim.y;
    
#define VAR(y, x) domain_variable[(y)*(N+2)+(x)]
    //The corners can't be run in parallel, so it does not
    //matter if we check them with if-statements, all other
    //threads will either stall or do the same operation.
    //Nonetheless, it will take 4 cycles (of writing data).
    if (x_idx == 0 && y_idx == 0)
        VAR(0, 0) = sign * VAR(2, 2);
    if (x_idx == 0 && y_idx == y_dim_stride - 1)
        VAR(N + 1, 0) = sign * VAR(N - 1, 2);
    if (x_idx == x_dim_stride - 1 && y_idx == 0)
        VAR(0, N + 1) = sign * VAR(2, N - 1);
    if (x_idx == x_dim_stride - 1 && y_idx == y_dim_stride - 1)
        VAR(N + 1, N + 1) = sign * VAR(N - 1, N - 1);
    
    
    
    //  Under follows an example illustration of how the
    //  boundary conditions are set by the different threads.
    //
    //  X-dim is horizontal
    //  Y-dim is vertical
    //  Cell ((x, y)) (0,0) is in the lower left.
    //  The x's mark a 4x4 grid of blocks that we
    //  can run on the card simultaneously.
    //  The 8x8 grid is the data we want to
    //  operate on.
    //
    //  The x's that are along the border (left and down)
    //  will be responsible for calculating the boundary for
    //  the boundary at their location.
    //
    //  However, we need to do the rest of the left and bottom sides.
    //  Also, the entire right and top side. The thread at (1,0) will be
    //  responsible for (1 + x_stride, 0) = (5,0). The same thing happens
    //  For the left side in the y-dim.
    //
    //  Lastly the top side of the 4x4 will be responsible for the top side
    //  of the 8x8. This by doing as the bottom, just offsetting to the top cells.
    //  +--+--+--+--+--+--+--+--+
    //  |  |  |  |  |  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  |  |  |  |  |  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  |  |  |  |  |  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  |  |  |  |  |  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  | x| x| x| x|  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  | x| x| x| x|  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  | x| x| x| x|  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //  | x| x| x| x|  |  |  |  |
    //  +--+--+--+--+--+--+--+--+
    //
    //  This table shows the thread index that is responsible for each boundary in the grid.
    //  +--+--+--+--+--+--+--+--+
    //  |12|13|14|15|12|13|14|15|
    //  +--+--+--+--+--+--+--+--+
    //  | 8|  |  |  |  |  |  |11|
    //  +--+--+--+--+--+--+--+--+
    //  | 4|  |  |  |  |  |  | 7|
    //  +--+--+--+--+--+--+--+--+
    //  | 0|  |  |  |  |  |  | 3|
    //  +--+--+--+--+--+--+--+--+
    //  |12|13|14|15|  |  |  |15|
    //  +--+--+--+--+--+--+--+--+
    //  | 8| 9|10|11|  |  |  |11|
    //  +--+--+--+--+--+--+--+--+
    //  | 4| 5| 6| 7|  |  |  | 7|
    //  +--+--+--+--+--+--+--+--+
    //  | 0| 1| 2| 3| 0| 1| 2| 3|
    //  +--+--+--+--+--+--+--+--+
    
    if (x_idx == 0) //At edge of thread grid, left
        for (int_t y = y_idx; y <= N; y += y_dim_stride)
            if (y != 0) //We don't want to overwrite the corner value with an incorrect value
                VAR(y, 0) = sign * VAR(y, 2);
    if (x_idx == x_dim_stride - 1) //At edge of thread grid, right
        for (int_t y = y_idx; y <= N; y += y_dim_stride)
            if (y != 0) //We don't want to overwrite the corner value with an incorrect value
                VAR(y, N + 1) = sign * VAR(y, N - 1);
    if (y_idx == 0) //At edge of thread grid, bottom
        for (int_t x = x_idx; x <= N; x += x_dim_stride)
            if (x != 0) //We don't want to overwrite the corner value with an incorrect value
                VAR(0, x) = sign * VAR(2, x);
    if (y_idx == y_dim_stride - 1) //At edge of thread grid, top
        for (int_t x = x_idx; x <= N; x++)
            if (x != 0) //We don't want to overwrite the corner value with an incorrect value
                VAR(N + 1, x) = sign * VAR(N - 1, x);

#undef VAR
}


void domain_init(void)
{
    int elements = (N + 2) * (N + 2);
    
    // TODO: Allocate device buffers for masses, velocities and accelerations.
    hipMalloc(&d_mass_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_1, elements * sizeof(real_t));
    
    hipMalloc(&d_mass_velocity_x_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_x_1, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_1, elements * sizeof(real_t));
    
    hipMalloc(&d_mass_velocity, elements * sizeof(real_t));
    
    hipMalloc(&d_velocity_x, elements * sizeof(real_t));
    hipMalloc(&d_velocity_y, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_x, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_y, elements * sizeof(real_t));
    // -----------------------------------------------------
    h_mass_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_1 = (real_t *) calloc(elements, sizeof(real_t));
    
    h_mass_velocity_x_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_x_1 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_1 = (real_t *) calloc(elements, sizeof(real_t));
    
    h_mass_velocity = (real_t *) calloc(elements, sizeof(real_t));
    
    h_velocity_x = (real_t *) calloc(elements, sizeof(real_t));
    h_velocity_y = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_x = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_y = (real_t *) calloc(elements, sizeof(real_t));
    
    for (int_t y = 1; y <= N; y++)
    {
        for (int_t x = 1; x <= N; x++)
        {
            h_mass_0[y * (N + 2) + x] = 1e-3;
            h_mass_velocity_x_0[y * (N + 2) + x] = 0.0;
            h_mass_velocity_y_0[y * (N + 2) + x] = 0.0;
            
            real_t cx = x - N / 2;
            real_t cy = y - N / 2;
            if (sqrt(cx * cx + cy * cy) < N / 20.0)
            {
                h_mass_0[y * (N + 2) + x] -= 5e-4 * exp(
                        -4 * pow(cx, 2.0) / (real_t) (N)
                        - 4 * pow(cy, 2.0) / (real_t) (N)
                );
            }
            
            h_mass_0[y * (N + 2) + x] *= density;
        }
    }
    
    dx = domain_size / (real_t) N;
    dt = 5e-2;
    
    //Transfer the host-side data to the device
    //****************** CUDA ******************
    hipMemcpy(d_mass_0, h_mass_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_1, h_mass_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_0, h_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_1, h_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_mass_velocity, h_mass_velocity, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_velocity_x, h_velocity_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_y, h_velocity_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, h_acceleration_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_y, h_acceleration_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
    //****************** CUDA ******************
}


void domain_save(int_t iteration)
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset(filename, 0, 256 * sizeof(char));
    sprintf(filename, "data/%.5ld.bin", index);
    
    FILE *out = fopen(filename, "wb");
    if (!out)
    {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(1);
    }
    //fwrite ( mass[0], (N+2)*(N+2), sizeof(real_t), out );
    for (int_t y = 1; y <= N; y++)
    {
        fwrite(&h_mass_0[y * (N + 2) + 1], N, sizeof(real_t), out);
    }
    fclose(out);
}

void domain_finalize(void)
{
    free(h_mass_0);
    free(h_mass_1);
    free(h_mass_velocity_x_0);
    free(h_mass_velocity_x_1);
    free(h_mass_velocity_y_0);
    free(h_mass_velocity_y_1);
    free(h_mass_velocity);
    free(h_velocity_x);
    free(h_velocity_y);
    free(h_acceleration_x);
    free(h_acceleration_y);
    
    // TODO: Free device arrays
    hipFree(d_mass_0);
    hipFree(d_mass_1);
    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_mass_velocity_y_0);
    hipFree(d_mass_velocity_y_1);
    hipFree(d_mass_velocity);
    hipFree(d_velocity_x);
    hipFree(d_velocity_y);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
}
