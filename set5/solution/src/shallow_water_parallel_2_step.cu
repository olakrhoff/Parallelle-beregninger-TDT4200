#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// TDT4200 Parallel Computing - Graded CUDA
// ---------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <unistd.h>
#include <math.h>

#include "../inc/argument_utils.h"


typedef int64_t int_t;
typedef double real_t;

int_t
        N,
        max_iteration,
        snapshot_frequency;

const real_t
        domain_size = 10.0,
        gravity = 9.81,
        density = 997.0;

real_t
        *h_mass_0 = NULL,
        *h_mass_1 = NULL,
        *d_mass_0 = NULL,
        *d_mass_1 = NULL,
        
        *h_mass_velocity_x_0 = NULL,
        *h_mass_velocity_x_1 = NULL,
        *d_mass_velocity_x_0 = NULL,
        *d_mass_velocity_x_1 = NULL,
        
        *h_mass_velocity_y_0 = NULL,
        *h_mass_velocity_y_1 = NULL,
        *d_mass_velocity_y_0 = NULL,
        *d_mass_velocity_y_1 = NULL,
        
        *h_mass_velocity = NULL,
        *d_mass_velocity = NULL,
        
        *h_velocity_x = NULL,
        *d_velocity_x = NULL,
        *h_velocity_y = NULL,
        *d_velocity_y = NULL,
        
        *h_acceleration_x = NULL,
        *d_acceleration_x = NULL,
        *h_acceleration_y = NULL,
        *d_acceleration_y = NULL,
        dx,
        dt;

#define PN(y, x)         mass_0[(y)*(N+2)+(x)]
#define PN_next(y, x)    mass_1[(y)*(N+2)+(x)]
#define PNU(y, x)        mass_velocity_x_0[(y)*(N+2)+(x)]
#define PNU_next(y, x)   mass_velocity_x_1[(y)*(N+2)+(x)]
#define PNV(y, x)        mass_velocity_y_0[(y)*(N+2)+(x)]
#define PNV_next(y, x)   mass_velocity_y_1[(y)*(N+2)+(x)]
#define PNUV(y, x)       mass_velocity[(y)*(N+2)+(x)]
#define U(y, x)          velocity_x[(y)*(N+2)+(x)]
#define V(y, x)          velocity_y[(y)*(N+2)+(x)]
#define DU(y, x)         acceleration_x[(y)*(N+2)+(x)]
#define DV(y, x)         acceleration_y[(y)*(N+2)+(x)]


#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/*
void time_step(real_t *velocity_x, real_t *velocity_y,
               real_t *acceleration_x, real_t *acceleration_y,
               real_t *mass_velocity_x_0, real_t *mass_velocity_x_1,
               real_t *mass_velocity_y_0, real_t *mass_velocity_y_1,
               real_t *mass_velocity, real_t *mass_0, real_t *mass_1
);*/

void __global__ time_step_1(real_t *mass_0,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity,
                            real_t *velocity_x,
                            real_t *velocity_y,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N);

void __global__ time_step_2(real_t *mass_0,
                            real_t *mass_1,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_x_1,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity_y_1,
                            real_t *mass_velocity,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N,
                            real_t dx,
                            real_t dt);

// TODO: Rewrite boundary_condition as a device function.
void __device__ boundary_condition(real_t *domain_variable, int sign, int_t N, int_t x_idx, int_t y_idx);

void domain_init(void);

void domain_save(int_t iteration);

void domain_finalize(void);

// Pthreads threaded domain save function
void *domain_save_threaded(void *iter);

void swap(real_t **t1, real_t **t2)
{
    real_t *tmp;
    tmp = *t1;
    *t1 = *t2;
    *t2 = tmp;
}


int main(int argc, char **argv)
{
    
    OPTIONS *options = parse_args(argc, argv);
    if (!options)
    {
        fprintf(stderr, "Argument parsing failed\n");
        exit(1);
    }
    
    N = options->N;
    max_iteration = options->max_iteration;
    snapshot_frequency = options->snapshot_frequency;
    
    domain_init();
    
    //We want 1024 threads per block (maximum) for a square this is 32x32 = 1024
    //Hence we have 32 threads in x and y dim, in thread block
    dim3 threadBlockDims = {32, 32, 1};
    //We want to divide the 2D space into equal sized squares making a grid.
    //This grid then needs to be N / 32(number of threads in dim in thread block) (rounded up).
    //Therefore, we get ceiling(N / threadBlock.x) and corresponding for y-dim.
    dim3 gridDims = {(uint32_t)((N + 2) / threadBlockDims.x) + (((N + 2) % threadBlockDims.x) != 0),
                     (uint32_t)((N + 2) / threadBlockDims.y) + (((N + 2) % threadBlockDims.y) != 0),
                     1};
    
    for (int_t iteration = 0; iteration <= max_iteration; iteration++)
    {
        /*
        time_step(h_velocity_x, h_velocity_y,
                  h_acceleration_x, h_acceleration_y,
                  h_mass_velocity_x_0, h_mass_velocity_x_1,
                  h_mass_velocity_y_0, h_mass_velocity_y_1,
                  h_mass_velocity, h_mass_0, h_mass_1
        );*/
        
        // TODO: Launch time_step kernels
        time_step_1<<<gridDims, threadBlockDims>>>(d_mass_0,
                              d_mass_velocity_x_0,
                              d_mass_velocity_y_0,
                              d_mass_velocity,
                              d_velocity_x,
                              d_velocity_y,
                              d_acceleration_x,
                              d_acceleration_y,
                              N);
        
        hipDeviceSynchronize();
        time_step_2<<<gridDims, threadBlockDims>>>(d_mass_0,
                              d_mass_1,
                              d_mass_velocity_x_0,
                              d_mass_velocity_x_1,
                              d_mass_velocity_y_0,
                              d_mass_velocity_y_1,
                              d_mass_velocity,
                              d_acceleration_x,
                              d_acceleration_y,
                              N,
                              dx,
                              dt);
        
        if (iteration % snapshot_frequency == 0)
        {
            printf(
                    "Iteration %ld of %ld, (%.2lf%% complete)\n",
                    iteration,
                    max_iteration,
                    100.0 * (real_t) iteration / (real_t) max_iteration
            );
            
            
            // TODO: Copy the masses from the device to host prior to domain_save
            //Transfer the device-side data to the host
            //****************** CUDA ******************
            int elements = (N + 2) * (N + 2);
            
            hipMemcpy(h_mass_0, d_mass_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_1, d_mass_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(h_mass_velocity_x_0, d_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_x_1, d_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_y_0, d_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_mass_velocity_y_1, d_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(h_mass_velocity, d_mass_velocity, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            
            hipMemcpy(h_velocity_x, d_velocity_x, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_velocity_y, d_velocity_y, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_acceleration_x, d_acceleration_x, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            hipMemcpy(h_acceleration_y, d_acceleration_y, elements * sizeof(real_t), hipMemcpyDeviceToHost);
            //****************** CUDA ******************
            
            domain_save(iteration);
        }
        
        // TODO: Swap device buffer pointers between iterations
        swap(&d_mass_0, &d_mass_1);
        swap(&d_mass_velocity_x_0, &d_mass_velocity_x_1);
        swap(&d_mass_velocity_y_0, &d_mass_velocity_y_1);
        
        swap(&h_mass_0, &h_mass_1);
        swap(&h_mass_velocity_x_0, &h_mass_velocity_x_1);
        swap(&h_mass_velocity_y_0, &h_mass_velocity_y_1);
    }
    
    domain_finalize();
    
    exit(EXIT_SUCCESS);
}


// TODO: Rewrite this function as one or more CUDA kernels
// ---------------------------------------------------------
// To ensure correct results, the participating threads in the thread
// grid must be synchronized after calculating the accelerations (DU, DV).
// If the grid is not synchronized, data dependencies cannot be guaranteed.
/*
void time_step(real_t *velocity_x, real_t *velocity_y,
               real_t *acceleration_x, real_t *acceleration_y,
               real_t *mass_velocity_x_0, real_t *mass_velocity_x_1,
               real_t *mass_velocity_y_0, real_t *mass_velocity_y_1,
               real_t *mass_velocity, real_t *mass_0, real_t *mass_1)
{
    boundary_condition(mass_0, 1);
    boundary_condition(mass_velocity_x_0, -1);
    boundary_condition(mass_velocity_y_0, -1);
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            U(y, x) = PNU(y, x) / PN(y, x);
            V(y, x) = PNV(y, x) / PN(y, x);
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNUV(y, x) = PN(y, x) * U(y, x) * V(y, x);
        }
    
    for (int_t y = 0; y <= N + 1; y++)
        for (int_t x = 0; x <= N + 1; x++)
        {
            DU(y, x) = PN(y, x) * U(y, x) * U(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
            DV(y, x) = PN(y, x) * V(y, x) * V(y, x)
                       + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNU_next(y, x) = 0.5 * (PNU(y, x + 1) + PNU(y, x - 1)) - dt * (
                    (DU(y, x + 1) - DU(y, x - 1)) / (2 * dx)
                    + (PNUV(y, x + 1) - PNUV(y, x - 1)) / (2 * dx)
            );
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PNV_next(y, x) = 0.5 * (PNV(y + 1, x) + PNV(y - 1, x)) - dt * (
                    (DV(y + 1, x) - DV(y - 1, x)) / (2 * dx)
                    + (PNUV(y + 1, x) - PNUV(y - 1, x)) / (2 * dx)
            );
        }
    
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {
            PN_next(y, x) = 0.25 * (PN(y, x + 1) + PN(y, x - 1) + PN(y + 1, x) + PN(y - 1, x)) - dt * (
                    (PNU(y, x + 1) - PNU(y, x - 1)) / (2 * dx)
                    + (PNV(y + 1, x) - PNV(y - 1, x)) / (2 * dx)
            );
        }
}*/

void __global__ time_step_1(real_t *mass_0,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity,
                            real_t *velocity_x,
                            real_t *velocity_y,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N)
{
    int global_index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (global_index_x >= N + 2 || global_index_y >= N + 2)
        return;
    
    
    int_t y = global_index_y;
    int_t x = global_index_x;
    
    boundary_condition(mass_0, 1, N, x, y);
    boundary_condition(mass_velocity_x_0, -1, N, x, y);
    boundary_condition(mass_velocity_y_0, -1, N, x, y);
    
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    U(y, x) = PNU(y, x) / PN(y, x);
    V(y, x) = PNV(y, x) / PN(y, x);
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNUV(y, x) = PN(y, x) * U(y, x) * V(y, x);
    /*}*/
    /*
    for (int_t y = 0; y <= N + 1; y++)
        for (int_t x = 0; x <= N + 1; x++)
        {*/
    DU(y, x) = PN(y, x) * U(y, x) * U(y, x) + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
    DV(y, x) = PN(y, x) * V(y, x) * V(y, x) + 0.5 * gravity * (PN(y, x) * PN(y, x) / density);
    /*}*/
}

void __global__ time_step_2(real_t *mass_0,
                            real_t *mass_1,
                            real_t *mass_velocity_x_0,
                            real_t *mass_velocity_x_1,
                            real_t *mass_velocity_y_0,
                            real_t *mass_velocity_y_1,
                            real_t *mass_velocity,
                            real_t *acceleration_x,
                            real_t *acceleration_y,
                            int_t N,
                            real_t dx,
                            real_t dt)
{
    int global_index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (global_index_x >= N + 2 || global_index_y >= N + 2)
        return;
    
    
    int_t y = global_index_y;
    int_t x = global_index_x;
    
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNU_next(y, x) = 0.5 * (PNU(y, x + 1) + PNU(y, x - 1)) - dt * ((DU(y, x + 1) - DU(y, x - 1)) / (2 * dx) + (PNUV(y, x + 1) - PNUV(y, x - 1)) / (2 * dx));
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PNV_next(y, x) = 0.5 * (PNV(y + 1, x) + PNV(y - 1, x)) - dt * ((DV(y + 1, x) - DV(y - 1, x)) / (2 * dx) + (PNUV(y + 1, x) - PNUV(y - 1, x)) / (2 * dx));
    /*}*/
    /*
    for (int_t y = 1; y <= N; y++)
        for (int_t x = 1; x <= N; x++)
        {*/
    PN_next(y, x) = 0.25 * (PN(y, x + 1) + PN(y, x - 1) + PN(y + 1, x) + PN(y - 1, x)) - dt * ((PNU(y, x + 1) - PNU(y, x - 1)) / (2 * dx) + (PNV(y + 1, x) - PNV(y - 1, x)) / (2 * dx));
    /*}*/
}

// TODO: Rewrite boundary_condition as a device function.
void __device__ boundary_condition(real_t *domain_variable, int sign, int_t N, int_t x_idx, int_t y_idx)
{
#define VAR(y, x) domain_variable[(y)*(N+2)+(x)]
    //if (x_idx == 0 && y_idx == 0)
        VAR(0, 0) = sign * VAR(2, 2);
    //if (x_idx == 0 && y_idx == N - 1)
        VAR(N + 1, 0) = sign * VAR(N - 1, 2);
    //if (x_idx == N - 1 && y_idx == 0)
        VAR(0, N + 1) = sign * VAR(2, N - 1);
    //if (x_idx == N - 1 && y_idx == N - 1)
        VAR(N + 1, N + 1) = sign * VAR(N - 1, N - 1);
    
    //if (x_idx == 0)
        for (int_t y = 1; y <= N; y++)
                VAR(y, 0) = sign * VAR(y, 2);
    //if (x_idx == N - 1)
        for (int_t y = 1; y <= N; y++)
            VAR(y, N + 1) = sign * VAR(y, N - 1);
    //if (y_idx == 0)
        for (int_t x = 1; x <= N; x++)
            VAR(0, x) = sign * VAR(2, x);
    //if (y_idx == N - 1)
        for (int_t x = 1; x <= N; x++)
            VAR(N + 1, x) = sign * VAR(N - 1, x);
        
#undef VAR
}


void domain_init(void)
{
    int elements = (N + 2) * (N + 2);
    
    // TODO: Allocate device buffers for masses, velocities and accelerations.
    hipMalloc(&d_mass_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_1, elements * sizeof(real_t));
    
    hipMalloc(&d_mass_velocity_x_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_x_1, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_0, elements * sizeof(real_t));
    hipMalloc(&d_mass_velocity_y_1, elements * sizeof(real_t));
    
    hipMalloc(&d_mass_velocity, elements * sizeof(real_t));
    
    hipMalloc(&d_velocity_x, elements * sizeof(real_t));
    hipMalloc(&d_velocity_y, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_x, elements * sizeof(real_t));
    hipMalloc(&d_acceleration_y, elements * sizeof(real_t));
    // -----------------------------------------------------
    h_mass_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_1 = (real_t *) calloc(elements, sizeof(real_t));
    
    h_mass_velocity_x_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_x_1 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_0 = (real_t *) calloc(elements, sizeof(real_t));
    h_mass_velocity_y_1 = (real_t *) calloc(elements, sizeof(real_t));
    
    h_mass_velocity = (real_t *) calloc(elements, sizeof(real_t));
    
    h_velocity_x = (real_t *) calloc(elements, sizeof(real_t));
    h_velocity_y = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_x = (real_t *) calloc(elements, sizeof(real_t));
    h_acceleration_y = (real_t *) calloc(elements, sizeof(real_t));
    
    for (int_t y = 1; y <= N; y++)
    {
        for (int_t x = 1; x <= N; x++)
        {
            h_mass_0[y * (N + 2) + x] = 1e-3;
            h_mass_velocity_x_0[y * (N + 2) + x] = 0.0;
            h_mass_velocity_y_0[y * (N + 2) + x] = 0.0;
            
            real_t cx = x - N / 2;
            real_t cy = y - N / 2;
            if (sqrt(cx * cx + cy * cy) < N / 20.0)
            {
                h_mass_0[y * (N + 2) + x] -= 5e-4 * exp(
                        -4 * pow(cx, 2.0) / (real_t) (N)
                        - 4 * pow(cy, 2.0) / (real_t) (N)
                );
            }
            
            h_mass_0[y * (N + 2) + x] *= density;
        }
    }
    
    dx = domain_size / (real_t) N;
    dt = 5e-2;
    
    //Transfer the host-side data to the device
    //****************** CUDA ******************
    hipMemcpy(d_mass_0, h_mass_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_1, h_mass_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_mass_velocity_x_0, h_mass_velocity_x_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_x_1, h_mass_velocity_x_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_0, h_mass_velocity_y_0, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass_velocity_y_1, h_mass_velocity_y_1, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_mass_velocity, h_mass_velocity, elements * sizeof(real_t), hipMemcpyHostToDevice);
    
    hipMemcpy(d_velocity_x, h_velocity_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_velocity_y, h_velocity_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_x, h_acceleration_x, elements * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(d_acceleration_y, h_acceleration_y, elements * sizeof(real_t), hipMemcpyHostToDevice);
    //****************** CUDA ******************
}


void domain_save(int_t iteration)
{
    int_t index = iteration / snapshot_frequency;
    char filename[256];
    memset(filename, 0, 256 * sizeof(char));
    sprintf(filename, "data/%.5ld.bin", index);
    
    FILE *out = fopen(filename, "wb");
    if (!out)
    {
        fprintf(stderr, "Failed to open file %s\n", filename);
        exit(1);
    }
    //fwrite ( mass[0], (N+2)*(N+2), sizeof(real_t), out );
    for (int_t y = 1; y <= N; y++)
    {
        fwrite(&h_mass_0[y * (N + 2) + 1], N, sizeof(real_t), out);
    }
    fclose(out);
}

void domain_finalize(void)
{
    free(h_mass_0);
    free(h_mass_1);
    free(h_mass_velocity_x_0);
    free(h_mass_velocity_x_1);
    free(h_mass_velocity_y_0);
    free(h_mass_velocity_y_1);
    free(h_mass_velocity);
    free(h_velocity_x);
    free(h_velocity_y);
    free(h_acceleration_x);
    free(h_acceleration_y);
    
    // TODO: Free device arrays
    hipFree(d_mass_0);
    hipFree(d_mass_1);
    hipFree(d_mass_velocity_x_0);
    hipFree(d_mass_velocity_x_1);
    hipFree(d_mass_velocity_y_0);
    hipFree(d_mass_velocity_y_1);
    hipFree(d_mass_velocity);
    hipFree(d_velocity_x);
    hipFree(d_velocity_y);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
}
